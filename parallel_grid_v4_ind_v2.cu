#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>

using namespace std;

__constant__ double accuracy[1];
__constant__ double bounds[4];
__constant__ int stride[1];
struct rectangle{
	double x1;
	double x2;
	double y1;
	double y2;
};

__host__ __device__ double g1(double x1,double x2 ,double l1_max){
return(x1*x1 + x2*x2 - l1_max*l1_max);
}
__host__ __device__ double g2(double x1,double x2 ,double l1_min){
return(l1_min*l1_min - x1*x1 - x2*x2);
}

__host__ __device__ double g3(double x1,double x2 ,double l2_max,double l0){
return((x1-l0)*(x1-l0) + x2*x2 - l2_max*l2_max);
}
__host__ __device__ double g4(double x1,double x2 ,double l2_min,double l0){
return(l2_min*l2_min  - (x1-l0)*(x1-l0) - x2*x2);
}
__host__ __device__ double interval_eval(double x1,double x2,double y1, double y2, double l, double l0, double lmax){
	double mm,mm1,mm2,mm3,mm4;
	double MM,MM1,MM2,MM3,MM4;
	mm1 = min(g1(x1,0,lmax),g1(x2,0,lmax)) + min(g1(lmax,y1,lmax),g1(lmax,y2,lmax));
	MM1 = max(g1(x1,0,lmax),g1(x2,0,lmax)) + max(g1(lmax,y1,lmax),g1(lmax,y2,lmax));

	mm2 = min(g2(x1,0,l),g2(x2,0,l)) + min(g2(l,y1,l),g2(l,y2,l));
	MM2 = max(g2(x1,0,l),g2(x2,0,l)) + max(g2(l,y1,l),g2(l,y2,l));

	mm3 = min(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + min(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));
	MM3 = max(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + max(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));

	mm4 = min(g4(x1,l,l,l0),g4(x2,l,l,l0)) + min(g4(l0,y1,l,l0),g4(l0,y2,l,l0));
	MM4 = max(g4(x1,l,l,l0),g4(x2,l,l,l0)) + max(g4(l0,y1,l,l0),g4(l0,y2,l,l0));

	mm = max(max(mm1,mm2),max(mm3,mm4));
	MM = max(max(MM1,MM2),max(MM3,MM4));
	char a = char(bool(MM < 0) + bool(mm < 0));

	return a;
}

__global__ void kernel(char* device_grid) {
double x1[100],x2[100],y1[100],y2[100];
long int i = blockIdx.x * blockDim.x + threadIdx.x;

for(int j = 0;j < stride[0];j++){

	x1[j] = bounds[0] + (threadIdx.x*stride[0]+j)*accuracy[0];
	x2[j] = bounds[0] + (threadIdx.x*stride[0]+j+1)*accuracy[0];
	y1[j] = bounds[2] + (blockIdx.x)*accuracy[0];
	y2[j] = bounds[2] + (blockIdx.x+1)*accuracy[0];

	double l = 8;
	double lmax = 12;
	double l0 = 5;

	device_grid[i*stride[0]+j] = interval_eval(x1[j],x2[j],y1[j],y2[j],l,l0,lmax);

	}

}



int main(){
  int l = 8;
	int l0 = 5;
	double approximation = 0.001;
	double lmax;
	lmax = l*1.5;
  cout<<fixed;
	cout.precision(3);
	rectangle r1;
	r1 = {-15,15,0,15};
  double const_bounds[4] = {r1.x1,r1.x2,r1.y1,r1.y2};
	int n_of_blocks = (r1.y2-r1.y1)/approximation;
	int n_of_threads = (r1.x2-r1.x1)/approximation;
  int offset[1] = {10};
	if(n_of_threads > 1000){
		offset[0] *=10;
		//n_of_threads /= 10;

	}
  cout<<"# <<<"<<n_of_blocks<<" , "<<n_of_threads<<">>>\n";
char* device_grid = new char[n_of_blocks * n_of_threads];

  std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();

	hipMallocManaged(&device_grid, n_of_blocks * n_of_threads * sizeof (int));

  hipMemcpyToSymbol(HIP_SYMBOL(accuracy), &approximation, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(stride), &offset, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(bounds), &const_bounds, 4*sizeof(double));
	cout<<"#"<<n_of_blocks<<"  "<<n_of_threads/offset[0]<<"\n";
	kernel<<<n_of_blocks,n_of_threads/offset[0]>>>(device_grid);
	hipDeviceSynchronize();

end = std::chrono:: high_resolution_clock::now();
int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
std::time_t end_time = std::chrono::system_clock::to_time_t(end);
cout<< "#. Время выполнения: " << elapsed_seconds << "  microseconds\n";
cout<<"#"<<n_of_blocks*n_of_threads<<"\n";
////BOUNDARY
//for(int i = 0; i < n_of_blocks*n_of_threads; i++){
	//if(int(device_grid[i])==1){
			//cout<<"["<<r1.x1+i%(n_of_threads)*approximation<<":"<<r1.x1+(i%(n_of_threads) + 1)*approximation<<"]:";
			//cout<<"["<<r1.y1+i/(n_of_threads)*approximation<<":"<<r1.y1+(i/(n_of_threads) + 1)*approximation<<"]\n";
	//}
//}
//// //internal
   //cout<<"_+_+_+_\n";
 //for(int i = 0; i < n_of_blocks*n_of_threads; i++){
 	//if(int(device_grid[i])==2){
 			//cout<<"["<<r1.x1+i%(n_of_threads)*approximation<<":"<<r1.x1+(i%(n_of_threads) + 1)*approximation<<"]:";
 			//cout<<"["<<r1.y1+i/(n_of_threads)*approximation<<":"<<r1.y1+(i/(n_of_threads) + 1)*approximation<<"]\n";
 	//}
 //}
int sq = 0;
for(int i = 0;i < n_of_blocks*n_of_threads;i++){
		if(int(device_grid[i]) == 2){
				sq++;
		}else{
			if(sq > 0){
					i-=sq;
					cout<<"["<<r1.x1+i%(n_of_threads)*approximation<<":"<<r1.x1+(i%(n_of_threads) + sq)*approximation<<"]:";
					cout<<"["<<r1.y1+i/(n_of_threads)*approximation<<":"<<r1.y1+(i/(n_of_threads) + 1)*approximation<<"]\n";
					i += sq;
					sq=0;
				}
			}
}


hipFree(device_grid);
}
