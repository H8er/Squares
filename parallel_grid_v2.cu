#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <math.h>
#include <vector>
using namespace std;

/*
double min(double a, double b){
	return(a<b?a:b);
}
double max(double a, double b){
	return(a>b?a:b);
}
*/
__constant__ double accuracy[1];
__constant__ double bounds[4];
struct rectangle{
	double x1;
	double x2;
	double y1;
	double y2;
	double d;
	double mg[4],Mg[4],mm,MM;
	double h,l;
};

__host__ __device__ double g1(double x1,double x2 ,double l1_max){
return(x1*x1 + x2*x2 - l1_max*l1_max);
}
__host__ __device__ double g2(double x1,double x2 ,double l1_min){
return(l1_min*l1_min - x1*x1 - x2*x2);
}

__host__ __device__ double g3(double x1,double x2 ,double l2_max,double l0){
return((x1-l0)*(x1-l0) + x2*x2 - l2_max*l2_max);
}
__host__ __device__ double g4(double x1,double x2 ,double l2_min,double l0){
return(l2_min*l2_min  - (x1-l0)*(x1-l0) - x2*x2);
}

__global__ void kernel(double* device_grid) {
double x1,x2,y1,y2;
int i = blockIdx.x * blockDim.x + threadIdx.x;
x1 = bounds[0]+(threadIdx.x)*accuracy[0];
x2 = bounds[0]+(threadIdx.x+1)*accuracy[0];
y1 = bounds[2]+(blockIdx.x)*accuracy[0];
y2 = bounds[2]+(blockIdx.x+1)*accuracy[0];
__syncwarp();
double mm,mm1,mm2,mm3,mm4;
double MM,MM1,MM2,MM3,MM4;
double l = 8;
double lmax = 12;
double l0 = 5;
double dia;

mm1 = min(g1(x1,0,lmax),g1(x2,0,lmax)) + min(g1(lmax,y1,lmax),g1(lmax,y2,lmax));
MM1 = max(g1(x1,0,lmax),g1(x2,0,lmax)) + max(g1(lmax,y1,lmax),g1(lmax,y2,lmax));

mm2 = min(g2(x1,0,l),g2(x2,0,l)) + min(g2(l,y1,l),g2(l,y2,l));
MM2 = max(g2(x1,0,l),g2(x2,0,l)) + max(g2(l,y1,l),g2(l,y2,l));

mm3 = min(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + min(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));
MM3 = max(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + max(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));

mm4 = min(g4(x1,l,l,l0),g4(x2,l,l,l0)) + min(g4(l0,y1,l,l0),g4(l0,y2,l,l0));
MM4 = max(g4(x1,l,l,l0),g4(x2,l,l,l0)) + max(g4(l0,y1,l,l0),g4(l0,y2,l,l0));

mm = max(max(mm1,mm2),max(mm3,mm4));
MM = max(max(MM1,MM2),max(MM3,MM4));

dia = sqrt(abs(x2-x1)*abs(x2-x1) + abs(y2-y1)*abs(y2-y1));
if(MM >= 0){
	if(mm > 0){
		if(((x1 <= 0 and x2 >= 0)and(y1<=0 and y2 >= lmax))or
((x1 <= l0 and x2 >= l0)and(y1<=0 and y2 >= lmax)))
			{
				device_grid[i] = x1;
				device_grid[i+blockDim.x*gridDim.x] = x2;
				device_grid[i+blockDim.x*gridDim.x*2] = y1;
				device_grid[i+blockDim.x*gridDim.x*3] = y2;
			}
		}
	else{
		device_grid[i] = x1;
		device_grid[i+blockDim.x*gridDim.x] = x2;
		device_grid[i+blockDim.x*gridDim.x*2] = y1;
		device_grid[i+blockDim.x*gridDim.x*3] = y2;
	}
	}
	else{
		//
		device_grid[i] = x1;
		device_grid[i+blockDim.x*gridDim.x] = x2;
		device_grid[i+blockDim.x*gridDim.x*2] = y1;
		device_grid[i+blockDim.x*gridDim.x*3] = y2;
	}

}



int main(){
  int l = 8;
	int l0 = 5;
	double approximation = 0.1;

	double lmax;
	lmax = l*1.5;
  cout<<fixed;
	cout.precision(3);
	//float t1,t2;
	rectangle r1;
	//r1 = {xmin-5, xmax+5, min(ymin,h)-5, ymax+5};
	r1 = {-15,15,0,15};
	r1.d = sqrt(abs(r1.x2-r1.x1)*abs(r1.x2-r1.x1) + abs(r1.y2-r1.y1)*abs(r1.y2-r1.y1));
  double const_bounds[4] = {r1.x1,r1.x2,r1.y1,r1.y2};
	//std::chrono::time_point<std::chrono:: high_resolution_clock> start, end; start = std::chrono::high_resolution_clock::now();
	int n_of_blocks = (r1.y2-r1.y1)/approximation;
	int n_of_threads = (r1.x2-r1.x1)/approximation;

	//double* host_grid = new double[n_of_blocks*n_of_threads*4];
	double* device_grid;

  std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();

	hipMallocManaged(&device_grid, n_of_blocks * n_of_threads * 4 * sizeof (double));


  hipMemcpyToSymbol(HIP_SYMBOL(accuracy), &approximation, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(bounds), &const_bounds, 4*sizeof(double));

	//double N_of_squares = n_of_blocks*n_of_threads;
  //
	kernel<<<n_of_blocks,n_of_threads>>>(device_grid);
	//cudaMemcpy (host_grid, device_grid, n_of_blocks * n_of_threads * 4 * sizeof (double), cudaMemcpyDeviceToHost);
hipDeviceSynchronize();

end = std::chrono:: high_resolution_clock::now();
int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
std::time_t end_time = std::chrono::system_clock::to_time_t(end);
cout<< "#. Время выполнения: " << elapsed_seconds << "  microseconds\n";

for(int i = 0; i<n_of_blocks*n_of_threads;i++){
	if((device_grid[i]!=device_grid[i+n_of_blocks*n_of_threads]) and (device_grid[i+n_of_blocks*n_of_threads*2]!=device_grid[i+n_of_blocks*n_of_threads*3])){
	cout<<"["<<device_grid[i]<<":"<<device_grid[i+n_of_blocks*n_of_threads]<<"]:["<<device_grid[i+n_of_blocks*n_of_threads*2]<<":"<<device_grid[i+n_of_blocks*n_of_threads*3]<<"]\n";
}
}

}
