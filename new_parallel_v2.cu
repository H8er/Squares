#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>

using namespace std;

__constant__ double accuracy[1];
__constant__ double bounds[4];
__constant__ int stride[1];
struct rectangle{
	double x1;
	double x2;
	double y1;
	double y2;
};

__host__ __device__ double g1(double x1,double x2 ,double l1_max){
return(x1*x1 + x2*x2 - l1_max*l1_max);
}
__host__ __device__ double g2(double x1,double x2 ,double l1_min){
return(l1_min*l1_min - x1*x1 - x2*x2);
}

__host__ __device__ double g3(double x1,double x2 ,double l2_max,double l0){
return((x1-l0)*(x1-l0) + x2*x2 - l2_max*l2_max);
}
__host__ __device__ double g4(double x1,double x2 ,double l2_min,double l0){
return(l2_min*l2_min  - (x1-l0)*(x1-l0) - x2*x2);
}
__forceinline__ __host__ __device__ char interval_eval(double x1,double x2,double y1, double y2, double l, double l0, double lmax){


	double mm,mm1,mm2,mm3,mm4;
	double MM,MM1,MM2,MM3,MM4;


	if(x1<0 and x2>0){
		 x1 = 0;
		 x2 = 0;
	}
	if(x1<l0 and x2>l0){
		 x1 = l0;
		 x2 = l0;
	}


	mm1 = min(g1(x1,0,lmax),g1(x2,0,lmax)) + min(g1(lmax,y1,lmax),g1(lmax,y2,lmax));
	MM1 = max(g1(x1,0,lmax),g1(x2,0,lmax)) + max(g1(lmax,y1,lmax),g1(lmax,y2,lmax));

	mm2 = min(g2(x1,0,l),g2(x2,0,l)) + min(g2(l,y1,l),g2(l,y2,l));
	MM2 = max(g2(x1,0,l),g2(x2,0,l)) + max(g2(l,y1,l),g2(l,y2,l));

	mm3 = min(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + min(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));
	MM3 = max(g3(x1,lmax,lmax,l0),g3(x2,lmax,lmax,l0)) + max(g3(l0,y1,lmax,l0),g3(l0,y2,lmax,l0));

	mm4 = min(g4(x1,l,l,l0),g4(x2,l,l,l0)) + min(g4(l0,y1,l,l0),g4(l0,y2,l,l0));
	MM4 = max(g4(x1,l,l,l0),g4(x2,l,l,l0)) + max(g4(l0,y1,l,l0),g4(l0,y2,l,l0));

	mm = max(max(mm1,mm2),max(mm3,mm4));
	MM = max(max(MM1,MM2),max(MM3,MM4));
	char a = char(bool(MM <= 0) + bool(mm <= 0));

	return a;
}

__global__ void kernel(char* device_grid) {
double x1[100],x2[100],y1[100],y2[100];
long int i = blockIdx.x * blockDim.x + threadIdx.x;

for(int j = 0;j < stride[0];j++){

	x1[j] = bounds[0] + (threadIdx.x*stride[0]+j)*accuracy[0];
	x2[j] = bounds[0] + (threadIdx.x*stride[0]+j+1)*accuracy[0];
	y1[j] = bounds[2] + (blockIdx.x)*accuracy[0];
	y2[j] = bounds[2] + (blockIdx.x+1)*accuracy[0];

	double l = 8;
	double lmax = 12;
	double l0 = 5;

	device_grid[i*stride[0]+j] = interval_eval(x1[j],x2[j],y1[j],y2[j],l,l0,lmax);
	}

}



int main(){
  int l = 8;
	int l0 = 5;
	double approximation = 0.01;
	double lmax;
	lmax = l*1.5;
  cout<<fixed;
	cout.precision(4);
	rectangle r1;

	long int elapsed_seconds = 0;
	double step = 0.5;
	for(double x_c = -15; x_c < 15;x_c += 2*step){
		int b_s = 0;
		for(double y_c = 0;y_c < 15; y_c += step){
			std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
			start = std::chrono::high_resolution_clock::now();
			r1.x1 = x_c;
			r1.x2 = x_c + 2*step;
			r1.y1 = y_c;
			r1.y2 = y_c + step;

				if(int(interval_eval(r1.x1,r1.x2,r1.y1,r1.y2,l,l0,lmax))!=0){
					if(int(interval_eval(r1.x1,r1.x2,r1.y1,r1.y2,l,l0,lmax))==2){
						// cout<<"_+_+_+_\n";
						// cout<<"["<<r1.x1<<":"<<r1.x2<<"]:";
						// cout<<"["<<r1.y1<<":"<<r1.y2<<"]\n";
						// cout<<"_+_+_+_\n";
						b_s++;
					}else{
						if(b_s > 0){
							cout<<"_+_+_+_\n";
							cout<<"["<<r1.x1<<":"<<r1.x2<<"]:";
							cout<<"["<<r1.y1-(b_s)*step<<":"<<r1.y2-step<<"]\n";
							cout<<"_+_+_+_\n";
							b_s = 0;
						}
					  double const_bounds[4] = {r1.x1,r1.x2,r1.y1,r1.y2};
						int n_of_blocks = (r1.y2-r1.y1)/approximation;
						int n_of_threads = (r1.x2-r1.x1)/approximation;
					  int offset[1] = {10};
							if(n_of_threads > 1000){
								offset[0] *=10;
							}
					char* device_grid = new char[n_of_blocks * n_of_threads * offset[0]];
					hipMallocManaged(&device_grid, n_of_blocks * n_of_threads * sizeof (char));
				  hipMemcpyToSymbol(HIP_SYMBOL(accuracy), &approximation, sizeof(double));
				  hipMemcpyToSymbol(HIP_SYMBOL(stride), &offset, sizeof(int));
				  hipMemcpyToSymbol(HIP_SYMBOL(bounds), &const_bounds, 4*sizeof(double));

					kernel<<<n_of_blocks,n_of_threads/offset[0]>>>(device_grid);
					hipDeviceSynchronize();

					end = std::chrono:: high_resolution_clock::now();
					int t = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
					elapsed_seconds += std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
					std::time_t end_time = std::chrono::system_clock::to_time_t(end);
					//BOUNDARY
					for(int i = 0; i < n_of_blocks*n_of_threads; i++){
						if(int(device_grid[i])==1){
								cout<<"["<<r1.x1+i%(n_of_threads)*approximation<<":"<<r1.x1+(i%(n_of_threads) + 1)*approximation<<"]:";
								cout<<"["<<r1.y1+i/(n_of_threads)*approximation<<":"<<r1.y1+(i/(n_of_threads) + 1)*approximation<<"]\n";
						}
					}
					//internal
					 // for(int i = 0; i < n_of_blocks*n_of_threads; i++){
					 // 	if(int(device_grid[i])==2){
					 // 			cout<<"["<<r1.x1+i%(n_of_threads)*approximation<<":"<<r1.x1+(i%(n_of_threads) + 1)*approximation<<"]:";
					 // 			cout<<"["<<r1.y1+i/(n_of_threads)*approximation<<":"<<r1.y1+(i/(n_of_threads) + 1)*approximation<<"]\n";
					 // 	}
					 // }
					cout<<"_+_+_+_\n";

					int sq = 0;
					int lq = 1;

					for(int j = 0;j < n_of_blocks;j++){
						inner_cycle:
						sq=0;
						for(int i = 0;i < n_of_threads;i++){
							if((int(device_grid[j*n_of_threads+i]) == 2)){
								sq++;
								if(i==n_of_threads-1){
									if(sq!=n_of_threads){
										i++;
										i-=sq;
										cout<<"["<<r1.x1+i*approximation<<":"<<r1.x1+(i + sq)*approximation<<"]:";
										cout<<"["<<r1.y1+j*approximation<<":"<<r1.y1+(j + 1)*approximation<<"]\n";
										i += sq;
										sq = 0;
									}
									else{
										lq++;
										j++;
										goto inner_cycle;
									}
								}
							}
							else{
								if(sq > 0){
										i-=sq;
										cout<<"["<<r1.x1+i*approximation<<":"<<r1.x1+(i + sq)*approximation<<"]:";
										cout<<"["<<r1.y1+j*approximation<<":"<<r1.y1+(j + 1)*approximation<<"]\n";
										i += sq;
										sq = 0;
										if(lq>1){
											cout<<"["<<r1.x1<<":"<<r1.x1+n_of_threads*approximation<<"]:";
											cout<<"["<<r1.y1+(j-lq+1)*approximation<<":"<<r1.y1+(j)*approximation<<"]\n";
											lq=1;
										}
								}
								else{
									if(lq>1){
										cout<<"["<<r1.x1<<":"<<r1.x1+n_of_threads*approximation<<"]:";
										cout<<"["<<r1.y1+(j-lq+1)*approximation<<":"<<r1.y1+(j)*approximation<<"]\n";
										lq=1;
									}
								}
							}
						}
					}
					cout<<"_+_+_+_\n";
					hipFree(device_grid);
					}
				}
		}
}

cout<< "#. Время выполнения: " << elapsed_seconds << "  microseconds\n";
return 0;
}
