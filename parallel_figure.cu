#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <math.h>
#include <vector>
using namespace std;

/*
double min(double a, double b){
	return(a<b?a:b);
}
double max(double a, double b){
	return(a>b?a:b);
}
*/
struct rectangle{
	double x1;
	double x2;
	double y1;
	double y2;
	double d;
	double mg[4],Mg[4],mm,MM;
	double h,l;
};

__host__ __device__ double g1(double x1,double x2 ,double l1_max){
return(x1*x1 + x2*x2 - l1_max*l1_max);
}
__host__ __device__ double g2(double x1,double x2 ,double l1_min){
return(l1_min*l1_min - x1*x1 - x2*x2);
}

__host__ __device__ double g3(double x1,double x2 ,double l2_max,double l0){
return((x1-l0)*(x1-l0) + x2*x2 - l2_max*l2_max);
}
__host__ __device__ double g4(double x1,double x2 ,double l2_min,double l0){
return(l2_min*l2_min  - (x1-l0)*(x1-l0) - x2*x2);
}

__global__ void kernel(double xmin,double xmax, double ymin, double ymax, double d,double* global_grid) {
__shared__ double x1[1536];
__shared__ double x2[1536];
__shared__ double y1[1536];
__shared__ double y2[1536];
int i = blockIdx.x * blockDim.x + threadIdx.x;
x1[i] = xmin+(threadIdx.x)*(abs(xmax-xmin)/32.);
x2[i] = xmin+(threadIdx.x+1)*(abs(xmax-xmin)/32.);
y1[i] = ymax-(blockIdx.x+1)*(abs(ymax-ymin)/48.);
y2[i] = ymax-(blockIdx.x)*(abs(ymax-ymin)/48.);
__syncwarp();
double mm,mm1,mm2,mm3,mm4;
double MM,MM1,MM2,MM3,MM4;
double l = 8;
double lmax = 12;
double l0 = 5;
double dia;
d = 1;
mm1 = min(g1(x1[i],0,lmax),g1(x2[i],0,lmax)) + min(g1(lmax,y1[i],lmax),g1(lmax,y2[i],lmax));
MM1 = max(g1(x1[i],0,lmax),g1(x2[i],0,lmax)) + max(g1(lmax,y1[i],lmax),g1(lmax,y2[i],lmax));

mm2 = min(g2(x1[i],0,l),g2(x2[i],0,l)) + min(g2(l,y1[i],l),g2(l,y2[i],l));
MM2 = max(g2(x1[i],0,l),g2(x2[i],0,l)) + max(g2(l,y1[i],l),g2(l,y2[i],l));

mm3 = min(g3(x1[i],lmax,lmax,l0),g3(x2[i],lmax,lmax,l0)) + min(g3(l0,y1[i],lmax,l0),g3(l0,y2[i],lmax,l0));
MM3 = max(g3(x1[i],lmax,lmax,l0),g3(x2[i],lmax,lmax,l0)) + max(g3(l0,y1[i],lmax,l0),g3(l0,y2[i],lmax,l0));

mm4 = min(g4(x1[i],l,l,l0),g4(x2[i],l,l,l0)) + min(g4(l0,y1[i],l,l0),g4(l0,y2[i],l,l0));
MM4 = max(g4(x1[i],l,l,l0),g4(x2[i],l,l,l0)) + max(g4(l0,y1[i],l,l0),g4(l0,y2[i],l,l0));

mm = max(max(mm1,mm2),max(mm3,mm4));
MM = max(max(MM1,MM2),max(MM3,MM4));

dia = sqrt(abs(x2[i]-x1[i])*abs(x2[i]-x1[i]) + abs(y2[i]-y1[i])*abs(y2[i]-y1[i]));
if(MM >= 0){
	if(mm > 0){
		if(((x1[i] <= 0 and x2[i] >= 0)and(y1[i]<=0 and y2[i] >= lmax))or
((x1[i] <= l0 and x2[i] >= l0)and(y1[i]<=0 and y2[i] >= lmax)))
		{////cout<<"aaaaaaaaaaaaa\n";
			if(dia <= d){
				//
				global_grid[i] = x1[i];
				global_grid[i+1536] = x2[i];
				global_grid[i+1536*2] = y1[i];
				global_grid[i+1536*3] = y2[i];
			}
			else{
				//division(VectorOfRectangles);
			}
		}
	}
	else{
		if(dia <= d){
			//
			global_grid[i] = x1[i];
			global_grid[i+1536] = x2[i];
			global_grid[i+1536*2] = y1[i];
			global_grid[i+1536*3] = y2[i];
		}
		else{
			//division(VectorOfRectangles);
		}
	}
	}
	else{
		//
		global_grid[i] = x1[i];
		global_grid[i+1536] = x2[i];
		global_grid[i+1536*2] = y1[i];
		global_grid[i+1536*3] = y2[i];
	}



}






bool restriction_1(double x1,double l1_max, double l2_max, double l0){
	return((x1 >= -l1_max) and (x1 <= l0+l2_max));
}
bool restriction_2(double x2,double l1_max, double l2_max){
	return((x2>=0) and (x2 <= min(l1_max,l2_max)));
}

vector<rectangle> division(vector<rectangle> &VectorOfRectangles){
rectangle temp = VectorOfRectangles.front();
rectangle r1;
rectangle r2;
double l = 1;
double lmax = 4;
double l0 = 5;
temp.h = abs(temp.y2 - temp.y1);
temp.l = abs(temp.x2 - temp.x1);
if(temp.h > temp.l){
	 r1 = {temp.x1,temp.x2,temp.y1,temp.y1+temp.h/2.};
	 r2 = {temp.x1,temp.x2,temp.y1+temp.h/2.,temp.y2};

}
else{
	 r1 = {temp.x1,temp.x1+temp.l/2.,temp.y1,temp.y2};
	 r2 = {temp.x1+temp.l/2.,temp.x2,temp.y1,temp.y2};
}
r1.d = sqrt(abs(r1.x2-r1.x1)*abs(r1.x2-r1.x1) + abs(r1.y2-r1.y1)*abs(r1.y2-r1.y1));
r2.d = sqrt(abs(r2.x2-r2.x1)*abs(r2.x2-r2.x1) + abs(r2.y2-r2.y1)*abs(r2.y2-r2.y1));

r1.mg[0] = min(min(g1(r1.x1,r1.y1,lmax),g1(r1.x1,r1.y2,lmax)),min(g1(r1.x2,r1.y2,lmax),g1(r1.x2,r1.y1,lmax)));
r1.Mg[0] = max(max(g1(r1.x1,r1.y1,lmax),g1(r1.x1,r1.y2,lmax)),max(g1(r1.x2,r1.y2,lmax),g1(r1.x2,r1.y1,lmax)));
r1.mg[1] = min(min(g2(r1.x1,r1.y1,l),g2(r1.x1,r1.y2,l)),min(g2(r1.x2,r1.y2,l),g2(r1.x2,r1.y1,l)));
r1.Mg[1] = max(max(g2(r1.x1,r1.y1,l),g2(r1.x1,r1.y2,l)),max(g2(r1.x2,r1.y2,l),g2(r1.x2,r1.y1,l)));
r1.mm = max(r1.mg[0],r1.mg[1]);
r1.MM = max(r1.Mg[0],r1.Mg[1]);

r2.mg[0] = min(min(g1(r2.x1,r2.y1,lmax),g1(r2.x1,r2.y2,lmax)),min(g1(r2.x2,r2.y2,lmax),g1(r2.x2,r2.y1,lmax)));
r2.Mg[0] = max(max(g1(r2.x1,r2.y1,lmax),g1(r2.x1,r2.y2,lmax)),max(g1(r2.x2,r2.y2,lmax),g1(r2.x2,r2.y1,lmax)));
r2.mg[1] = min(min(g2(r2.x1,r2.y1,l),g2(r2.x1,r2.y2,l)),min(g2(r2.x2,r2.y2,l),g2(r2.x2,r2.y1,l)));
r2.Mg[1] = max(max(g2(r2.x1,r2.y1,l),g2(r2.x1,r2.y2,l)),max(g2(r2.x2,r2.y2,l),g2(r2.x2,r2.y1,l)));
r2.mm = max(r2.mg[0],r2.mg[1]);
r2.MM = max(r2.Mg[0],r2.Mg[1]);

VectorOfRectangles.push_back(r1);
VectorOfRectangles.push_back(r2);

	return VectorOfRectangles;
}

int main(){
	int l = 1;
	int l0 = 5;
	double approximation = 0.1;
	cout.precision(5);
	double lmax;
	lmax = 4;
	float t1,t2;
	rectangle r1;
	//r1 = {xmin-5, xmax+5, min(ymin,h)-5, ymax+5};
	r1 = {-15,15,0,15};
	r1.d = sqrt(abs(r1.x2-r1.x1)*abs(r1.x2-r1.x1) + abs(r1.y2-r1.y1)*abs(r1.y2-r1.y1));

	std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
			start = std::chrono::high_resolution_clock::now();

	double *host_grid = new double[1536*4];
	double* global_grid;
	hipMalloc ((void **) &global_grid, 1536*4 * sizeof (double));

	kernel<<<48,32>>>(r1.x1,r1.x2,r1.y1,r1.y2,approximation,global_grid);
	hipMemcpy (host_grid, global_grid, 1536*4 * sizeof (double), hipMemcpyDeviceToHost);


	end = std::chrono:: high_resolution_clock::now();

			int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
															 (end-start).count();
			std::time_t end_time = std::chrono::system_clock::to_time_t(end);

			//std:://cout<< "Время выполнения: " << elapsed_seconds << "  microseconds\n";
	t1 = elapsed_seconds;
start = std::chrono::high_resolution_clock::now();

	vector<rectangle> VectorOfRectangles;
	vector<rectangle> InternalRectangles;
	vector<rectangle> BoundRectangles;

	r1.mg[0] = min(min(g1(r1.x1,r1.y1,lmax),g1(r1.x1,r1.y2,lmax)),min(g1(r1.x2,r1.y2,lmax),g1(r1.x2,r1.y1,lmax)));
  r1.Mg[0] = max(max(g1(r1.x1,r1.y1,lmax),g1(r1.x1,r1.y2,lmax)),max(g1(r1.x2,r1.y2,lmax),g1(r1.x2,r1.y1,lmax)));
  r1.mg[1] = min(min(g2(r1.x1,r1.y1,l),g2(r1.x1,r1.y2,l)),min(g2(r1.x2,r1.y2,l),g2(r1.x2,r1.y1,l)));
  r1.Mg[1] = max(max(g2(r1.x1,r1.y1,l),g2(r1.x1,r1.y2,l)),max(g2(r1.x2,r1.y2,l),g2(r1.x2,r1.y1,l)));
  r1.mm = max(r1.mg[0],r1.mg[1]);
  r1.MM = max(r1.Mg[0],r1.Mg[1]);

	cout<<fixed;
	cout.precision(3);
	VectorOfRectangles.push_back(r1);


  while((VectorOfRectangles.size() > 0)){ //(VectorOfRectangles.size() < 10) and
    if(VectorOfRectangles.front().MM >= 0){
      if(VectorOfRectangles.front().mm > 0){
        if(VectorOfRectangles.front().x1 < 0 and VectorOfRectangles.front().x2 > 0){////cout<<"aaaaaaaaaaaaa\n";
          if(VectorOfRectangles.front().d <= approximation){
            BoundRectangles.push_back(VectorOfRectangles.front());
          }
          else{
            division(VectorOfRectangles);
          }
        }
      }
      else{
        if(VectorOfRectangles.front().d <= approximation){
          BoundRectangles.push_back(VectorOfRectangles.front());
        }
        else{
          division(VectorOfRectangles);
        }
      }
      }
      else{
        InternalRectangles.push_back(VectorOfRectangles.front());
      }
VectorOfRectangles.erase(VectorOfRectangles.begin(),VectorOfRectangles.begin()+1);
}
//cout<<VectorOfRectangles.size()<<" "<<InternalRectangles.size()<<" "<<BoundRectangles.size()<<"\n";

for(int i = 0;i<BoundRectangles.size();i++){
	//cout<<"["<<BoundRectangles[i].x1<<":"<<BoundRectangles[i].x2<<"]:[";	//cout<<BoundRectangles[i].y1<<":"<<BoundRectangles[i].y2<<"]\n";
}

for(int i = 0;i<InternalRectangles.size();i++){
	//cout<<"["<<InternalRectangles[i].x1<<":"<<InternalRectangles[i].x2<<"]:["<<InternalRectangles[i].y1<<":"<<InternalRectangles[i].y2<<"]\n";
}
//cout<<48*1024/sizeof(double)/4;

end = std::chrono:: high_resolution_clock::now();

		elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
														 (end-start).count();
		 end_time = std::chrono::system_clock::to_time_t(end);
t2 = elapsed_seconds;
//std:://cout<< "Время выполнения: " << elapsed_seconds << "  microseconds\n";

for(int i = 0; i<1536;i++){
	if(host_grid[i]!=host_grid[i+1536]){
	cout<<"["<<host_grid[i]<<":"<<host_grid[i+1536]<<"]:["<<host_grid[i+1536*2]<<":"<<host_grid[i+1536*3]<<"]\n";
}
}







}
