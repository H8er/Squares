#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <math.h>
#include <vector>
using namespace std;

/*
double min(double a, double b){
	return(a<b?a:b);
}
double max(double a, double b){
	return(a>b?a:b);
}
*/
__constant__ double accuracy[1];
__constant__ double bounds[4];
__constant__ int stride[1];
struct rectangle{
	double x1;
	double x2;
	double y1;
	double y2;
	double d;
	double mg[4],Mg[4],mm,MM;
	double h,l;
};

__host__ __device__ double g1(double x1,double x2 ,double l1_max){
return(x1*x1 + x2*x2 - l1_max*l1_max);
}
__host__ __device__ double g2(double x1,double x2 ,double l1_min){
return(l1_min*l1_min - x1*x1 - x2*x2);
}

__host__ __device__ double g3(double x1,double x2 ,double l2_max,double l0){
return((x1-l0)*(x1-l0) + x2*x2 - l2_max*l2_max);
}
__host__ __device__ double g4(double x1,double x2 ,double l2_min,double l0){
return(l2_min*l2_min  - (x1-l0)*(x1-l0) - x2*x2);
}

__global__ void kernel(double* device_grid) {
double x1[10],x2[10],y1[10],y2[10];
int i = blockIdx.x * blockDim.x + threadIdx.x;

for(int j = 0;j < stride[0];j++){

x1[j] = bounds[0] + (threadIdx.x*stride[0]+j)*accuracy[0];
x2[j] = bounds[0] + (threadIdx.x*stride[0]+j+1)*accuracy[0];
y1[j] = bounds[2] + (blockIdx.x)*accuracy[0];
y2[j] = bounds[2] + (blockIdx.x+1)*accuracy[0];

double mm,mm1,mm2,mm3,mm4;
double MM,MM1,MM2,MM3,MM4;
double l = 8;
double lmax = 12;
double l0 = 5;
double dia;

mm1 = min(g1(x1[j],0,lmax),g1(x2[j],0,lmax)) + min(g1(lmax,y1[j],lmax),g1(lmax,y2[j],lmax));
MM1 = max(g1(x1[j],0,lmax),g1(x2[j],0,lmax)) + max(g1(lmax,y1[j],lmax),g1(lmax,y2[j],lmax));

mm2 = min(g2(x1[j],0,l),g2(x2[j],0,l)) + min(g2(l,y1[j],l),g2(l,y2[j],l));
MM2 = max(g2(x1[j],0,l),g2(x2[j],0,l)) + max(g2(l,y1[j],l),g2(l,y2[j],l));

mm3 = min(g3(x1[j],lmax,lmax,l0),g3(x2[j],lmax,lmax,l0)) + min(g3(l0,y1[j],lmax,l0),g3(l0,y2[j],lmax,l0));
MM3 = max(g3(x1[j],lmax,lmax,l0),g3(x2[j],lmax,lmax,l0)) + max(g3(l0,y1[j],lmax,l0),g3(l0,y2[j],lmax,l0));

mm4 = min(g4(x1[j],l,l,l0),g4(x2[j],l,l,l0)) + min(g4(l0,y1[j],l,l0),g4(l0,y2[j],l,l0));
MM4 = max(g4(x1[j],l,l,l0),g4(x2[j],l,l,l0)) + max(g4(l0,y1[j],l,l0),g4(l0,y2[j],l,l0));

mm = max(max(mm1,mm2),max(mm3,mm4));
MM = max(max(MM1,MM2),max(MM3,MM4));

dia = sqrt(abs(x2[j]-x1[j])*abs(x2[j]-x1[j]) + abs(y2[j]-y1[j])*abs(y2[j]-y1[j]));
if(MM >= 0){
	if(mm > 0){
		if(((x1[j] <= 0 and x2[j] >= 0)and(y1[j]<=0 and y2[j] >= lmax))or
((x1[j] <= l0 and x2[j] >= l0)and(y1[j]<=0 and y2[j] >= lmax)))
			{
				device_grid[i*stride[0]+j] = x1[j];
			  device_grid[blockDim.x*gridDim.x*stride[0] + i*stride[0]+j] = x2[j];
			  device_grid[blockDim.x*gridDim.x*stride[0]*2 + i*stride[0]+j] = y1[j];
			  device_grid[blockDim.x*gridDim.x*stride[0]*3 + i*stride[0]+j] = y2[j];
			}
		}
	else{
		device_grid[i*stride[0]+j] = x1[j];
	  device_grid[blockDim.x*gridDim.x*stride[0] + i*stride[0]+j] = x2[j];
	  device_grid[blockDim.x*gridDim.x*stride[0]*2 + i*stride[0]+j] = y1[j];
	  device_grid[blockDim.x*gridDim.x*stride[0]*3 + i*stride[0]+j] = y2[j];
	}
	}
	else{
		//
		device_grid[i*stride[0]+j] = x1[j];
		device_grid[blockDim.x*gridDim.x*stride[0] + i*stride[0]+j] = x2[j];
		device_grid[blockDim.x*gridDim.x*stride[0]*2 + i*stride[0]+j] = y1[j];
		device_grid[blockDim.x*gridDim.x*stride[0]*3 + i*stride[0]+j] = y2[j];
	}

}


__syncwarp();

}



int main(){
  int l = 8;
	int l0 = 5;
	double approximation = 0.01;

	double lmax;
	lmax = l*1.5;
  cout<<fixed;
	cout.precision(3);
	//float t1,t2;
	rectangle r1;
	//r1 = {xmin-5, xmax+5, min(ymin,h)-5, ymax+5};
	r1 = {-15,15,0,15};
	r1.d = sqrt(abs(r1.x2-r1.x1)*abs(r1.x2-r1.x1) + abs(r1.y2-r1.y1)*abs(r1.y2-r1.y1));
  double const_bounds[4] = {r1.x1,r1.x2,r1.y1,r1.y2};
	//std::chrono::time_point<std::chrono:: high_resolution_clock> start, end; start = std::chrono::high_resolution_clock::now();
	int n_of_blocks = (r1.y2-r1.y1)/approximation;
	int n_of_threads = (r1.x2-r1.x1)/approximation;
  int offset[1] = {10};

  cout<<"# <<<"<<n_of_blocks<<" , "<<n_of_threads/offset[0]<<">>>\n";
	//double* host_grid = new double[n_of_blocks*n_of_threads*4];
	double* device_grid;

  std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();

	hipMallocManaged(&device_grid, n_of_blocks * n_of_threads * 4 * sizeof (double));


  hipMemcpyToSymbol(HIP_SYMBOL(accuracy), &approximation, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(stride), &offset, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(bounds), &const_bounds, 4*sizeof(double));

	//double N_of_squares = n_of_blocks*n_of_threads;
	//printf("%d\n",blockDim.x*gridDim.x*stride[0]);
  //
	kernel<<<n_of_blocks,n_of_threads/offset[0]>>>(device_grid);
	//cudaMemcpy (host_grid, device_grid, n_of_blocks * n_of_threads * 4 * sizeof (double), cudaMemcpyDeviceToHost);
hipDeviceSynchronize();

end = std::chrono:: high_resolution_clock::now();
int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
std::time_t end_time = std::chrono::system_clock::to_time_t(end);
cout<< "#. Время выполнения: " << elapsed_seconds << "  microseconds\n";
cout<<"#"<<n_of_blocks*n_of_threads<<"\n";
for(int i = 0; i < n_of_blocks*n_of_threads; i++){
	if((device_grid[i]!=device_grid[i+n_of_blocks*n_of_threads]) and (device_grid[i+n_of_blocks*n_of_threads*2]!=device_grid[i+n_of_blocks*n_of_threads*3])){
	cout<<"["<<device_grid[i]<<":"<<device_grid[i+n_of_blocks*n_of_threads]<<"]:["<<device_grid[i+n_of_blocks*n_of_threads*2]<<":"<<device_grid[i+n_of_blocks*n_of_threads*3]<<"]\n";
}
}

}
